#include "hip/hip_runtime.h"
#include <mpi.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <thread>
#include <chrono>
#ifndef NSYS
    #include "nvToolsExt.h"
#endif

// CUDA kernel function
__global__ void gpuKernel() {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Adding a delay (approx. 1 second)
    clock_t start = clock();
    while (clock() - start < 4000000000);
}

int main(int argc, char** argv) {
    // Initialize MPI
    MPI_Init(&argc, &argv);
    clock_t start;

    // Get the number of processes
    int numProcs;
    MPI_Comm_size(MPI_COMM_WORLD, &numProcs);

    // Get the rank of the process
    int rank;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);

    // Initialize CUDA for this process and set the device
    hipSetDevice(rank % numProcs);

    // Create CUDA stream
    hipStream_t stream;
    hipStreamCreate(&stream);

    // Allocate device memory
    int *d_data;
    hipMalloc(&d_data, sizeof(int) * 256);

    // Launch GPU kernel asynchronously
    double startTime = MPI_Wtime();
    gpuKernel<<<16, 16, 0, stream>>>();

    // Do CPU work concurrently
#ifndef NSYS
    nvtxRangePushA("CPU Code");
#endif
    double startTime2 = MPI_Wtime();
    // Simulate some CPU work with a 1 second delay
    printf("cpuFunction start\n");
    //start = clock();
    //while (clock() - start < 1000000);
    std::this_thread::sleep_for(std::chrono::seconds(1));
    std::cout << "CPU work done by rank " << rank << std::endl;
    double timeCPU = (MPI_Wtime() - startTime2);
#ifndef NSYS
    nvtxRangePop();
#endif

    // Wait for the GPU to finish
    hipStreamSynchronize(stream);
    //hipDeviceSynchronize();
    printf("GPU work done by rank %d\n", rank);
    printf("CPU Time %f\n",timeCPU);
    double timeCPUGPU = (MPI_Wtime() - startTime);
    printf("GPU Time %f\n",timeCPUGPU-timeCPU);
    printf("Total time: %f\n", timeCPUGPU);
    hipFree(d_data);
    hipStreamDestroy(stream);
    MPI_Finalize();
    return 0;
}
