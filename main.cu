#include "hip/hip_runtime.h"
/* Copyright (c) 1993-2015, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */
#include <mpi.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <thread>
#include <chrono>
#ifndef NSYS
    #include "nvToolsExt.h"
#endif

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
#endif
  return result;
}

__global__ void gpuKernel()
{
  clock_t start = clock();
  while (clock() - start < 400000000);
}

int main(int argc, char **argv)
{
  
  MPI_Init(&argc, &argv);
  // Get the number of processes
  int numProcs;
  MPI_Comm_size(MPI_COMM_WORLD, &numProcs);

  // Get the rank of the process
  int rank;
  MPI_Comm_rank(MPI_COMM_WORLD, &rank);
  
  //clock_t start;
  const int blockSize = 256, nStreams = 1;
  const int n = 4 * 1024 * blockSize * nStreams;
  const int streamSize = n / nStreams;
  const int streamBytes = streamSize * sizeof(float);
  const int bytes = n * sizeof(float);
   
  int devId = 0;
  if (argc > 1) devId = atoi(argv[1]);

  hipDeviceProp_t prop;
  checkCuda( hipGetDeviceProperties(&prop, devId));
  printf("Device : %s\n", prop.name);
  checkCuda( hipSetDevice(devId) );
  
  // allocate pinned host memory and device memory
  float *a, *d_a;
  checkCuda( hipHostMalloc((void**)&a, bytes) );      // host pinned
  checkCuda( hipMalloc((void**)&d_a, bytes) ); // device

  float ms; // elapsed time in milliseconds
  
  // create events and streams
  hipEvent_t startEvent, stopEvent, dummyEvent;
  hipStream_t stream[nStreams];
  checkCuda( hipEventCreate(&startEvent) );
  checkCuda( hipEventCreate(&stopEvent) );
  checkCuda( hipEventCreate(&dummyEvent) );
  for (int i = 0; i < nStreams; ++i)
    checkCuda( hipStreamCreate(&stream[i]) );
  
  // asynchronous version 2: 
  // loop over copy, loop over kernel, loop over copy
  memset(a, 0, bytes);
  checkCuda( hipEventRecord(startEvent,0) );
  for (int i = 0; i < 1; ++i)
  {
    int offset = i * streamSize;
    checkCuda( hipMemcpyAsync(&d_a[offset], &a[offset], 
                               streamBytes, hipMemcpyHostToDevice,
                               stream[i]) );
  }
  for (int i = 0; i < 1; ++i)
  {
    int offset = i * streamSize;
    gpuKernel<<<16, 16, 0, stream[i]>>>();
    hipStreamQuery(stream[i]);
  }
#ifndef NSYS
    nvtxRangePushA("CPU Code");
#endif
    std::this_thread::sleep_for(std::chrono::seconds(1));
#ifdef NSYS
    nvtxRangePop();
#endif
  for (int i = 0; i < 1; ++i)
  {
    int offset = i * streamSize;
    checkCuda( hipMemcpyAsync(&a[offset], &d_a[offset], 
                               streamBytes, hipMemcpyDeviceToHost,
                               stream[i]) );
    hipStreamQuery(stream[i]);
  }
  hipDeviceSynchronize();
  checkCuda( hipEventRecord(stopEvent, 0) );
  checkCuda( hipEventSynchronize(stopEvent) );
  checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
  printf("Time for asynchronous V2 transfer and execute (ms): %f\n", ms);
  // cleanup
  checkCuda( hipEventDestroy(startEvent) );
  checkCuda( hipEventDestroy(stopEvent) );
  checkCuda( hipEventDestroy(dummyEvent) );
  for (int i = 0; i < nStreams; ++i)
    checkCuda( hipStreamDestroy(stream[i]) );
  hipFree(d_a);
  hipHostFree(a);
  // Finalize MPI
  MPI_Finalize();

  return 0;
}